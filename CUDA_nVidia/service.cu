#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_timer.h>
#include "service.h"

extern "C"
void dev_alloc(int device, void** pparr, int sz)
{
	if(hipSetDevice(device))
		printf("ERROR Initializing device #%d\n",device);
	else
	{
		checkCudaErrors( hipMalloc( pparr, sz));
		hipDeviceSynchronize();
	}
}

extern "C"
void dev_h2d(int device, real* host_arr, real* dev_arr, int sz)
{
	//printf("\tcopying source of size %d from %p to %p\n", sz, host_arr, dev_arr);

	if(hipSetDevice(device))
		printf("ERROR Initializing device #%d\n",device);
	else
	{
		checkCudaErrors( hipMemcpy( dev_arr, host_arr, sz, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
	}
	//printf("\t\tdone\n");

}

extern "C"
void dev_d2h(int device, const real* dev_arr, real* host_arr, int sz)
{
	if(hipSetDevice(device))
		printf("ERROR Initializing device #%d\n",device);
	else
	{
		checkCudaErrors( hipMemcpy( host_arr, dev_arr, sz, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
	}
}

